#include <hip/hip_runtime.h>
#include "../src/C++/graph.cuh"
#include "../src/C++/edge.cuh"
#include "../src/C++/node.cuh"
#include "../src/C++/node_vector.cuh"
#include <stdio.h>

int main(){
    graph_t* graph = graph_load("graph1");
    node_t* new_node = node_init_gpu(0);
    node_add_attr(new_node, "name", "Matt");
    node_t* new_node2 = node_init_gpu(1);
    node_t* new_node3 = node_init_gpu(2);
    node_add_attr(new_node2, "name", "Lisa");
    node_add_attr(new_node3, "name", "Eric");
    graph_add_node(graph, new_node);
    graph_add_node(graph, new_node2);
    graph_add_node(graph, new_node3);
    edge_t* new_edge = edge_init_gpu(1, 0, 1);
    graph_add_edge(graph, new_edge);
    
    graph_find_V(graph, "name", "Matt");
    for(int i = 0; i < graph->nodes->size; i++){
	char* temp_name = node_get_attr(graph->nodes->stuff[i], "name");
	printf("%d is the id of %s\n", graph->nodes->stuff[i]->id, temp_name);
    }
    return 0;
}
