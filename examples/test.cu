#include <hip/hip_runtime.h>
#include "../src/C/graph.cuh"

int main(){
    graph_t* graph = graph_load("graph1");
    node_t* new_node = node_init(0);
    graph_add_node(graph, new_node);
    edge_t* new_edge = edge_init_gpu(1, 0, 1);
    graph_add_edge(graph, new_edge);
    return 0;
}
