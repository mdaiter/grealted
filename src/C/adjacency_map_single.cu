#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "adjacency_map_single.h"
#include <stdio.h>
#include "edge.h"

__global__ void adjacency_map_init_gpu(adjacency_map_t* map){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    int i = row * map->width + col;
    
    max(i, 0);
    min(i, map->width * map->height);

    map->connections[i] = 0;
}

__global__ void adjacency_map_connect_gpu(edge_t* edges, int num_edges, adjacency_map_t* map){
    
    int i = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x)*blockDim.x);

    max(i, 0);
    min(i, num_edges);

    int n_start = edges[i].n_start;
    int n_end = edges[i].n_end;
    
    int map_index = n_start * map->width + n_end;
    map->connections[map_index] = 1;
    printf("%d new value: %d\n", map_index, map->connections[map_index]);
}

adjacency_map_t* adjacency_map_init(int num_nodes, edge_t* edges, int num_edges){
    adjacency_map_t *map;// = (adjacency_map_t*)malloc(sizeof(adjacency_map_t));
    hipMallocManaged(&map, sizeof(adjacency_map_t));
    hipMallocManaged(&(map->connections), num_nodes * num_nodes * sizeof(int));
    //map->connections = (int*)malloc(sizeof(int) * num_nodes * num_nodes);

    map->width = num_nodes;
    map->height = num_nodes;

    map->stride = 0;
    
    //GPU stuff
//    adjacency_map_t *d_map;
//    int* d_connections;

//    hipMalloc((void**) &d_map, sizeof(adjacency_map_t));
//    hipMalloc((void**) &d_connections, num_nodes * num_nodes * sizeof(int));
    
//    hipMemcpy(d_map, map, sizeof(adjacency_map_t), hipMemcpyHostToDevice);
//    hipMemcpy(d_connections, map->connections, num_nodes * num_nodes, hipMemcpyHostToDevice);
    //hipMemcpy(&(d_map->connections), &d_connections, sizeof(int*), hipMemcpyHostToDevice);

//    edge_t* d_edges;
//    hipMalloc((void**) &d_edges, num_edges * sizeof(edge_t));
//    hipMemcpy(d_edges, edges, num_edges * sizeof(edge_t), hipMemcpyHostToDevice);

    adjacency_map_init_gpu<<<1, 3>>>(map);
    hipDeviceSynchronize();
    //adjacency_map_connect_gpu<<<1, 3>>>(edges, num_edges, map);
    
    hipDeviceSynchronize();

//    hipMemcpy(map, d_map, sizeof(adjacency_map_t), hipMemcpyDeviceToHost);
    //Synchronize everything
//    hipFree(map);
//    hipFree(edges);

    return map;

}

int main(){
    edge_t* edges;// = (edge_t*) malloc(sizeof(edge_t) * 3);
    hipMallocManaged(&edges, 3 * sizeof(edge_t));
    
    edges[0].id = 2;
    edges[0].n_start = 1 ;
    edges[0].n_end = 2;
    
    edges[1].id = 0;
    edges[1].n_start = 2;
    edges[1].n_end = 1;

    edges[2].id = 1;
    edges[2].n_start = 1;
    edges[2].n_end = 1;

    adjacency_map_t* map = adjacency_map_init(3, edges, 3);
    printf("[ %d %d %d\n%d %d %d\n%d %d %d]", map->connections[0], map->connections[1], map->connections[2], map->connections[3], map->connections[4], map->connections[5], map->connections[6], map->connections[7], map->connections[8]);
    return 0;
}

void adjacency_map_resize(adjacency_map_t* map, int new_size){
    map->width = new_size;
    map->height = new_size;

    map->connections = (int*) realloc(map->connections, new_size * new_size * sizeof(int));
}
