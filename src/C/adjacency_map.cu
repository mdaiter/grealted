#include "hip/hip_runtime.h"
#include "adjacency_map.cuh"
#include <stdlib.h>
#include <hip/hip_runtime.h>

void adjacency_map_init(adjacency_map* map, int row_num, int col_num){
    map = (adjacency_map*) malloc(sizeof(adjacency_map));

    kv_init(map->rows);
}

__global__ void adjacency_map_add(adjacency_map* map, int new_row_num, int new_col_num){
    int i = threadIdx.x + blockIdx.x * blockDim.x;

}
