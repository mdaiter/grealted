#include "hip/hip_runtime.h"
#include "graph.cuh"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

#include "redis_client.h"
#include "../../hiredis/hiredis.h"
#include "node.h"
#include "edge.h"

#ifdef __cplusplus
}
#endif

int string_length(char *s)
{
    int c = 0;
        
    while(*(s+c))
        c++;
           
    return c;
}

void graph_init(graph_t* graph, char* name){
    graph = (graph_t*) malloc(sizeof(graph_t));
    kv_init(graph->nodes);
    kv_init(graph->edges);
    graph->adjacency_map = (adjacency_map*) malloc(sizeof(adjacency_map));

    graph->name = (char*) malloc(string_length(name) * sizeof(char));

    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }
    
    redis_client_set_node_size(graph->redis_context, graph->name, 0);
    redis_client_set_edge_size(graph->redis_context, graph->name, 0);
}

graph_t* graph_load(char* name){
    //First we handle making the actual object
    graph_t* graph = (graph_t*) malloc(sizeof(graph_t));
    //Then we deal with the interior nodes
    kv_init(graph->nodes);
    kv_init(graph->edges);
    graph->adjacency_map = (adjacency_map*) malloc(sizeof(adjacency_map));

    graph->name = (char*) malloc(string_length(name) * sizeof(char));
    graph->name = name;

    //Initialize context to server
    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }

    int node_size =  redis_client_get_node_size(graph->redis_context, graph->name);
    int edge_size = redis_client_get_edge_size(graph->redis_context, graph->name);

    //Get the nodes from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < node_size; i++){
        node_t* node = node_init(i);
        node_load(node, graph->redis_context, graph->name);
        kv_push(node_t*, graph->nodes, node);
    }
    
    //Get the edges from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < edge_size; i++){
        edge_t* edge = edge_init(i, -1, -1);
        edge_load(edge, graph->redis_context, graph->name);
        kv_push(edge_t*, graph->edges, edge);
    }
    return graph;
}

void graph_add_node(graph_t* graph, node_t* node){
    node_save(node, graph->redis_context, graph->name);
    //Take care of database stuff. Then add to graph
    redisReply* reply;
    reply = (redisReply*)redisCommand(graph->redis_context, "INCR %s:nodes:size", graph->name);
    freeReplyObject(reply);

    //Now add to graph
    kv_push(node_t*, graph->nodes, node);
}

void graph_add_edge(graph_t* graph, edge_t* edge){
    edge_save(edge, graph->redis_context, graph->name);
    redisReply* reply;
    reply = (redisReply*)redisCommand(graph->redis_context, "INCR %s:edges:size", graph->name);
    freeReplyObject(reply);

    kv_push(edge_t*, graph->edges, edge);
}

void graph_add_edge_between_nodes(graph_t* graph, edge_t* edge, node_t* n_start, node_t* n_end){
    edge->n_start = n_start->id;
    edge->n_end = n_end->id;
    graph_add_edge(graph, edge);
}

__global__ void graph_find(node_t* nodes, char* key, void* value, node_t* d_node_out){
    __shared__ int d_node_out_size;
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;


    any_t holder = malloc(sizeof(*any_t));

    hashmap_get(nodes[i].attr, key, holder);

    if ( holder == value ){
        d_node_out[d_node_out_size] = nodes[i];
        atomicAdd(d_node_out_size);
        d_node_out = (node_t*) realloc(d_node_out, (d_node_out_size + 1) * sizeof(node_t));
    }
}

node_t* graph_find_V(graph_t* graph, char* key, any_t value){
    node_t* h_nodes = (node_t*)graph->nodes->a;

    node_t* d_nodes;
    char* d_key;
    any_t d_value;

    hipMalloc((void**)&d_nodes, kv_size(graph->nodes) * sizeof(node_t));
    hipMalloc((void**)&key, string_length(key) * sizeof(char));
    hipMalloc((void**)&value, sizeof(any_t));
    
    hipMemcpy(d_nodes, h_nodes, kv_size(graph->nodes) * sizeof(node_t), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, string_length(key) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_value, value, sizeof(any_t), hipMemcpyHostToDevice);

    graph_find<<<1, 1024>>>(d_nodes, key, value);


}
