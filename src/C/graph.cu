#include "graph.cuh"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

extern "C" {
#include "redis_client.h"
#include "../../hiredis/hiredis.h"
#include "node.h"
#include "edge.h"
}

int string_length(char *s)
{
    int c = 0;
        
    while(*(s+c))
        c++;
           
    return c;
}

void graph_init(graph_t* graph, char* name){
    //First we handle making the actual object
    graph = (graph_t*) malloc(sizeof(graph_t));
    //Then we deal with the interior nodes
    kv_init(graph->nodes);
    kv_init(graph->edges);
    graph->adjacency_map = (adjacency_map*) malloc(sizeof(adjacency_map));

    graph->name = (char*) malloc(string_length(name) * sizeof(char));
    graph->name = name;

    //Initialize context to server
    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }

    int node_size =  redis_client_get_node_size(graph->redis_context, graph->name);
    int edge_size = redis_client_get_edge_size(graph->redis_context, graph->name);

    //Get the nodes from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < node_size; i++){
        node_t* node;
        node_init(node, -1);
        node_load(node, graph->redis_context);
        kv_push(node_t*, graph->nodes, node);
    }
    
    //Get the edges from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < edge_size; i++){
        
    }
}


