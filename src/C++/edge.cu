#include "edge.cuh"
#include <hip/hip_runtime.h>

edge_t* edge_init_gpu(int _id, int _n_start, int _n_end){
	edge_t* e;
	hipMallocManaged(&e, sizeof(edge_t));
	
	e->id = _id;
	e->n_start = _n_start;
	e->n_end = _n_end;
	
	return e;
}


