#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "adjacency_map_single.cuh"
__global__ void adjacency_map_init_gpu(adjacency_map_t* map){
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    int i = row * map->width + col;
    
    max(i, 0);
    min(i, map->width * map->height);

    map->connections[i] = 0;
}

__global__ void adjacency_map_connect_gpu(edge_t** edges, int num_edges, adjacency_map_t* map){
    
    int i = threadIdx.x + (((gridDim.x * blockIdx.y) + blockIdx.x)*blockDim.x);

    max(i, 0);
    min(i, num_edges);

    int n_start = edges[i]->n_start;
    int n_end = edges[i]->n_end;
    
    int map_index = n_start * map->width + n_end;
    map->connections[map_index] = 1;
}

adjacency_map_t* adjacency_map_init(int num_nodes, edge_t** edges, int num_edges){
    adjacency_map_t *map;// = (adjacency_map_t*)malloc(sizeof(adjacency_map_t));
    hipMallocManaged(&map, sizeof(adjacency_map_t));
    hipMallocManaged(&(map->connections), num_nodes * num_nodes * sizeof(int));

    map->width = num_nodes;
    map->height = num_nodes;

    map->stride = 0;
    
    //GPU stuff

    adjacency_map_init_gpu<<<1, num_nodes * num_nodes>>>(map);
    hipDeviceSynchronize();
    adjacency_map_connect_gpu<<<1, num_edges>>>(edges, num_edges, map);
    
    hipDeviceSynchronize();

    return map;

}

void adjacency_map_resize(adjacency_map_t* map, int new_size){
    map->width = new_size;
    map->height = new_size;

    map->connections = (int*) realloc(map->connections, new_size * new_size * sizeof(int));
}
