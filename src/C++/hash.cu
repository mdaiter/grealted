#define _XOPEN_SOURCE 500 /* Enable certain library functions (strdup) on linux.  See feature_test_macros(7) */

#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include "hash.cuh"

__device__ __host__ int strlen(char* s){
	int c = 0;
	while(*(s+c)){
		c++;
	}
	return c;
}

__device__ __host__ int strcmp(char* str1, char* str2){
	if (str1 == NULL || str2 == NULL){
		return -1;
	}
	char* i = str1;
	char* j = str2;
	int i_len = strlen(str1);
	int j_len = strlen(str2);
	if (i_len != j_len){
		return -1;
	}
	
	for(int x = 0;  x < i_len && x < j_len; x++){
		if ((int)i[x] > (int)j[x]){
			return 1;
		}
		else if ((int)i[x] < (int)j[x]){
			return -1;
		}
		//i++;
		//j++;
	}
	return 0;
}


/* Create a new hashtable. */
hashtable_t *ht_create( int size ) {

	hashtable_t *hashtable = NULL;
	int i;

	if( size < 1 ) return NULL;

	/* Allocate the table itself. */
	hipMallocManaged(&hashtable, sizeof( hashtable_t ) );
	

	/* Allocate pointers to the head nodes. */
	hipMallocManaged( &(hashtable->table), sizeof( entry_t * ) * size );
	
	for( i = 0; i < size; i++ ) {
		hashtable->table[i] = NULL;
	}

	hashtable->size = size;

	return hashtable;	
}

/* Hash a string for a particular hash table. */
__device__ __host__ int ht_hash( hashtable_t *hashtable, char *key ) {

	unsigned long int hashval;
	int i = 0;

	/* Convert our string to an integer */
	while( hashval < ULONG_MAX && i < strlen( key ) ) {
		hashval = hashval << 8;
		hashval += key[ i ];
		i++;
	}

	return hashval % hashtable->size;
}

/* Create a key-value pair. */
entry_t *ht_newpair( char *key, char *value ) {
	entry_t *newpair;

	hipMallocManaged( &newpair, sizeof( entry_t ) );

	if( ( newpair->key = strdup( key ) ) == NULL ) {
		return NULL;
	}

	if( ( newpair->value = strdup( value ) ) == NULL ) {
		return NULL;
	}

	newpair->next = NULL;

	return newpair;
}

/* Insert a key-value pair into a hash table. */
void ht_set( hashtable_t *hashtable, char *key, char *value ) {
	int bin = 0;
	entry_t *newpair = NULL;
	entry_t *next = NULL;
	entry_t *last = NULL;

	bin = ht_hash( hashtable, key );

	next = hashtable->table[ bin ];

	while( next != NULL && next->key != NULL && strcmp( key, next->key ) > 0 ) {
		last = next;
		next = next->next;
	}

	/* There's already a pair.  Let's replace that string. */
	if( next != NULL && next->key != NULL && strcmp( key, next->key ) == 0 ) {

		hipFree( next->value );
		next->value = strdup( value );

	/* Nope, could't find it.  Time to grow a pair. */
	} else {
		newpair = ht_newpair( key, value );

		/* We're at the start of the linked list in this bin. */
		if( next == hashtable->table[ bin ] ) {
			newpair->next = next;
			hashtable->table[ bin ] = newpair;
	
		/* We're at the end of the linked list in this bin. */
		} else if ( next == NULL ) {
			last->next = newpair;
	
		/* We're in the middle of the list. */
		} else  {
			newpair->next = next;
			last->next = newpair;
		}
	}
}

/* Retrieve a key-value pair from a hash table. */
__device__ __host__ char *ht_get( hashtable_t *hashtable, char *key ) {
	int bin = 0;
	entry_t *pair;

	bin = ht_hash( hashtable, key );

	/* Step through the bin, looking for our value. */
	pair = hashtable->table[ bin ];
	while( pair != NULL && pair->key != NULL && strcmp( key, pair->key ) > 0 ) {
		pair = pair->next;
	}

	/* Did we actually find anything? */
	if( pair == NULL || pair->key == NULL || strcmp( key, pair->key ) != 0 ) {
		return NULL;

	} else {
		return pair->value;
	}
	
}


int main( int argc, char **argv ) {

	hashtable_t *hashtable = ht_create( 1 );

	ht_set( hashtable, "key1", "inky" );
	ht_set( hashtable, "key2", "pinky" );
	ht_set( hashtable, "key3", "blinky" );
	ht_set( hashtable, "key4", "floyd" );

	printf( "%s\n", ht_get( hashtable, "key1" ) );
	printf( "%s\n", ht_get( hashtable, "key2" ) );
	printf( "%s\n", ht_get( hashtable, "key3" ) );
	printf( "%s\n", ht_get( hashtable, "key4" ) );
	printf( "%d\n", hashtable->size);
	return 0;
}
