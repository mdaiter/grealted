#include "node_vector.cuh"
#include <hip/hip_runtime.h>

node_vector_t* node_vector_init(int size){
	node_vector_t* vec;
	hipMallocManaged(&vec, sizeof(node_vector_t));
	hipMallocManaged(&(vec->stuff), sizeof(node_t*) * size);
	vec->size = size;
	return vec;
}

void node_vector_add(node_vector_t* vec, node_t* new_element){
	node_t** vec2;
	hipMallocManaged(&vec2, sizeof(node_t*) * (vec->size + 1));
	hipMemcpy(vec2, vec->stuff, sizeof(node_t*) * (vec->size), hipMemcpyHostToHost);
	hipFree(vec->stuff);
	vec->stuff = vec2;
	vec->size++;
	vec->stuff[vec->size - 1] = new_element;
}

node_t** node_vector_raw_pointer(node_vector_t* vec){
	return vec->stuff;
}
