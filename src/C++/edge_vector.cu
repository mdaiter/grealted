#include "edge_vector.cuh"
#include <hip/hip_runtime.h>

edge_vector_t* edge_vector_init(int size){
	edge_vector_t* vec;
	hipMallocManaged(&vec, sizeof(edge_vector_t));
	hipMallocManaged(&(vec->stuff), sizeof(edge_t*) * size);
	vec->size = size;
	return vec;
}

void edge_vector_add(edge_vector_t* vec, edge_t* new_element){
	edge_t** vec2;
	hipMallocManaged(&vec2, sizeof(edge_t*) * (vec->size + 1));
	hipMemcpy(vec2, vec->stuff, sizeof(edge_t*) * (vec->size), hipMemcpyHostToHost);
	hipFree(vec->stuff);
	vec->stuff = vec2;
	vec->size++;
	vec->stuff[vec->size - 1] = new_element;
}

edge_t** edge_vector_raw_pointer(edge_vector_t* vec){
	return vec->stuff;
}
