#include "hip/hip_runtime.h"
#include "graph.cuh"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

#include "../C/redis_client.h"
#include "../../hiredis/hiredis.h"
#include "../C/node.h"
#include "../C/edge.h"

#ifdef __cplusplus
}
#endif

#include "node.cuh"
#include "edge.cuh"

int string_length(char *s)
{
    int c = 0;
        
    while(*(s+c))
        c++;
           
    return c;
}

void graph_init(graph_t* graph, char* name){
    graph = (graph_t*) malloc(sizeof(graph_t));
    graph->nodes = node_vector_init(0);
    graph->edges = edge_vector_init(0);
    graph->adjacency_map = (adjacency_map*) malloc(sizeof(adjacency_map));

    graph->name = (char*) malloc(string_length(name) * sizeof(char));

    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }
    
    redis_client_set_node_size(graph->redis_context, graph->name, 0);
    redis_client_set_edge_size(graph->redis_context, graph->name, 0);
}

graph_t* graph_load(char* name){
    //First we handle making the actual object
    graph_t* graph;
    hipMallocManaged(&graph, sizeof(graph_t));
    //Then we deal with the interior nodes
    graph->nodes = node_vector_init(0);
    graph->edges = edge_vector_init(0);

    graph->name = (char*) malloc(string_length(name) * sizeof(char));
    graph->name = name;

    //Initialize context to server
    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }

    int node_size =  redis_client_get_node_size(graph->redis_context, graph->name);
    int edge_size = redis_client_get_edge_size(graph->redis_context, graph->name);

    //Get the nodes from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < node_size; i++){
        node_t* node = node_init_gpu(i);
        node_load(node, graph->redis_context, graph->name);
        node_vector_add(graph->nodes, node);
    }
    
    //Get the edges from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < edge_size; i++){
        edge_t* edge = edge_init_gpu(i, -1, -1);
        edge_load(edge, graph->redis_context, graph->name);
        edge_vector_add(graph->edges, edge);
    }

    graph->adjacency_map = adjacency_map_init(node_size, graph->edges->stuff, edge_size);
    return graph;
}

void graph_add_node(graph_t* graph, node_t* node){
    node_save(node, graph->redis_context, graph->name);
    //Take care of database stuff. Then add to graph
    redisReply* reply;
    reply = (redisReply*)redisCommand(graph->redis_context, "INCR %s:nodes:size", graph->name);
    freeReplyObject(reply);

    //Now add to graph
    node_vector_add(graph->nodes, node);
}

void graph_add_edge(graph_t* graph, edge_t* edge){
    edge_save(edge, graph->redis_context, graph->name);
    redisReply* reply;
    reply = (redisReply*)redisCommand(graph->redis_context, "INCR %s:edges:size", graph->name);
    freeReplyObject(reply);

    edge_vector_add(graph->edges, edge);
}

void graph_add_edge_between_nodes(graph_t* graph, edge_t* edge, node_t* n_start, node_t* n_end){
    edge->n_start = n_start->id;
    edge->n_end = n_end->id;
    graph_add_edge(graph, edge);
}

__global__ void graph_find(node_vector_t* nodes, char* key, char* value, node_t* d_node_out){
    __shared__ node_vector_t* return_vector;
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    hashmap_get(nodes[i].attr, key, holder);

    if ( holder != NULL && holder == value ){
        d_node_out[d_node_out_size] = nodes[i];
        atomicAdd(d_node_out_size);
        d_node_out = (node_t*) realloc(d_node_out, (d_node_out_size + 1) * sizeof(node_t));
    }
}

node_t* graph_find_V(graph_t* graph, char* key, char* value){

    char* d_key;
    
    hipMalloc((void**)&key, string_length(key) * sizeof(char));
    
    hipMemcpy(d_key, key, string_length(key) * sizeof(char), hipMemcpyHostToDevice);
    node_vector_t* d_node_out = node_vector_init(0);
    graph_find<<<1, 1024>>>(graph->nodes, key, value, d_node_out);

    hipFree(d_key);
}
