#include "hip/hip_runtime.h"
#include "graph.cuh"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

#include "../C/redis_client.h"
#include "../../hiredis/hiredis.h"
#include "../C/node.h"
#include "../C/edge.h"

#ifdef __cplusplus
}
#endif

#include "hash.cuh"
#include "node.cuh"
#include "edge.cuh"
__device__ __host__ int strlen(char* s){
	int c = 0;
	while(*(s+c)){
		c++;
	}
	return c;
}

__device__ __host__ int strcmp(char* str1, char* str2){
	if (str1 == NULL || str2 == NULL){
		return -1;
	}
	char* i = str1;
	char* j = str2;
	int i_len = strlen(str1);
	int j_len = strlen(str2);
	if (i_len != j_len){
		return -1;
	}
	
	for(int x = 0;  x < i_len && x < j_len; x++){
		if ((int)i[x] > (int)j[x]){
			return 1;
		}
		else if ((int)i[x] < (int)j[x]){
			return -1;
		}
		//i++;
		//j++;
	}
	return 0;
}

/* Hash a string for a particular hash table. */
__device__ __host__ int ht_hash( hashtable_t *hashtable, char *key ) {

	unsigned long int hashval = 0;
	int i = 0;

	/* Convert our string to an integer */
	while( hashval < 0xFFFFFFFFUL && i < strlen( key ) ) {
		hashval = hashval << 8;
		hashval += key[ i ];
		i++;
	}
	printf("Hashval: %d\nhashtable->size: %d\n mod: %d\n", hashval, hashtable->size, hashval % hashtable->size);
	return hashval % hashtable->size;
}

/* Retrieve a key-value pair from a hash table. */
__device__ __host__ char *ht_get( hashtable_t *hashtable, char *key ) {
	int bin = 0;
	entry_t *pair;

	bin = ht_hash( hashtable, key );

	/* Step through the bin, looking for our value. */
	pair = hashtable->table[ bin ];
	printf("ht_get: %d for %d\n", pair == NULL, bin);
	while( pair != NULL && pair->key != NULL && strcmp( key, pair->key ) > 0 ) {
		printf("pair still null\n");
		pair = pair->next;
	}

	/* Did we actually find anything? */
	if( pair == NULL || pair->key == NULL || strcmp( key, pair->key ) != 0 ) {
		printf("Returning null\n");
		return NULL;

	} else {
		printf("Returned value!\n");
		return pair->value;
	}

}
void graph_init(graph_t* graph, char* name){
    graph = (graph_t*) malloc(sizeof(graph_t));
    graph->nodes = node_vector_init(0);
    graph->edges = edge_vector_init(0);
    graph->adjacency_map = (adjacency_map*) malloc(sizeof(adjacency_map));

    graph->name = (char*) malloc(strlen(name) * sizeof(char));

    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }
    
    redis_client_set_node_size(graph->redis_context, graph->name, 0);
    redis_client_set_edge_size(graph->redis_context, graph->name, 0);
}

graph_t* graph_load(char* name){
    //First we handle making the actual object
    graph_t* graph;
    hipMallocManaged(&graph, sizeof(graph_t));
    //Then we deal with the interior nodes
    graph->nodes = node_vector_init(0);
    graph->edges = edge_vector_init(0);

    graph->name = (char*) malloc(strlen(name) * sizeof(char));
    graph->name = name;

    //Initialize context to server
    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }

    int node_size =  redis_client_get_node_size(graph->redis_context, graph->name);
    int edge_size = redis_client_get_edge_size(graph->redis_context, graph->name);

    //Get the nodes from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < node_size; i++){
        //node_t* node = node_init_gpu(i);
        //node_load(node, graph->redis_context, graph->name);
        //node_vector_add(graph->nodes, node);
    }
    
    //Get the edges from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < edge_size; i++){
        //edge_t* edge = edge_init_gpu(i, -1, -1);
        //edge_load(edge, graph->redis_context, graph->name);
        //edge_vector_add(graph->edges, edge);
    }

    //graph->adjacency_map = adjacency_map_init(node_size, graph->edges->stuff, edge_size);
    return graph;
}

void graph_add_node(graph_t* graph, node_t* node){
    //node_save(node, graph->redis_context, graph->name);
    //Take care of database stuff. Then add to graph
    redisReply* reply;
    reply = (redisReply*)redisCommand(graph->redis_context, "INCR %s:nodes:size", graph->name);
    freeReplyObject(reply);

    //Now add to graph
    node_vector_add(graph->nodes, node);
}

void graph_add_edge(graph_t* graph, edge_t* edge){
    edge_save(edge, graph->redis_context, graph->name);
    redisReply* reply;
    reply = (redisReply*)redisCommand(graph->redis_context, "INCR %s:edges:size", graph->name);
    freeReplyObject(reply);

    edge_vector_add(graph->edges, edge);
}

void graph_add_edge_between_nodes(graph_t* graph, edge_t* edge, node_t* n_start, node_t* n_end){
    edge->n_start = n_start->id;
    edge->n_end = n_end->id;
    graph_add_edge(graph, edge);
}

__global__ void count(graph_t* graph, char* key, char* value, int* count){
	__shared__ int x;
	if (threadIdx.x + blockIdx.x * blockDim.x == 0) x = 0;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	char* holder = ht_get(graph->nodes->stuff[i]->attr, key);
	printf("%s\n", holder);
	if (strcmp(holder, value) == 0){
		printf("Found one\n");
		atomicAdd(&x, 1);
	}
	__syncthreads();
	printf("X: %d\n", x);
	count[0] = x;
}

__global__ void graph_find(graph_t* graph, char* key, char* value){
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    max(i, 0);
    min(i, graph->nodes->size);
	
    char* holder = ht_get(graph->nodes->stuff[i]->attr, key);
	/*
    if ( holder == NULL || !strcmp(holder, value) ){
    	graph->nodes->stuff[i]->is_selected = false;
    }*/
}

void graph_find_V(graph_t* graph, char* key, char* value){

    char* d_key;
    char* d_value;
    int* h_count = (int*) malloc(sizeof(int));
    h_count[0] = 0;
    int* d_count;    

    hipMalloc((void**)&d_key, strlen(key) * sizeof(char));
    hipMalloc((void**)&d_value, strlen(value) * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_key, key, strlen(key) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_value, value, strlen(value) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);

    count<<<1, 3>>>(graph, d_key, d_value, h_count);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    
    free(h_count);
    h_count = (int*) malloc(sizeof(int));
    hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Count: %d\n", h_count[0]);
    graph_find<<<1, 1024>>>(graph, d_key, d_value);
    hipDeviceSynchronize();
    hipFree(d_count);
    hipFree(d_key);
    hipFree(d_value);
}
