#include "hip/hip_runtime.h"
#include "graph.cuh"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

#include "../C/redis_client.h"
#include "../../hiredis/hiredis.h"
#include "../C/node.h"
#include "../C/edge.h"

#ifdef __cplusplus
}
#endif

#include "hash.cuh"
#include "node.cuh"
#include "edge.cuh"
__device__ __host__ int strlen(char* s){
	int c = 0;
	while(*(s+c)){
		c++;
	}
	return c;
}

__device__ __host__ int strcmp(char* str1, char* str2){
	if (str1 == NULL || str2 == NULL){
		return -1;
	}
	char* i = str1;
	char* j = str2;
	int i_len = strlen(str1);
	int j_len = strlen(str2);
	if (i_len != j_len){
		return -1;
	}
	
	for(int x = 0;  x < i_len && x < j_len; x++){
		if ((int)i[x] > (int)j[x]){
			return 1;
		}
		else if ((int)i[x] < (int)j[x]){
			return -1;
		}
		//i++;
		//j++;
	}
	return 0;
}

/* Hash a string for a particular hash table. */
__device__ __host__ int ht_hash( hashtable_t *hashtable, char *key ) {

	unsigned long int hashval;
	int i = 0;

	/* Convert our string to an integer */
	while( hashval < ULONG_MAX && i < strlen( key ) ) {
		hashval = hashval << 8;
		hashval += key[ i ];
		i++;
	}

	return hashval % hashtable->size;
}

/* Retrieve a key-value pair from a hash table. */
__device__ __host__ char *ht_get( hashtable_t *hashtable, char *key ) {
	int bin = 0;
	entry_t *pair;

	bin = ht_hash( hashtable, key );

	/* Step through the bin, looking for our value. */
	pair = hashtable->table[ bin ];
	while( pair != NULL && pair->key != NULL && strcmp( key, pair->key ) > 0 ) {
		pair = pair->next;
	}

	/* Did we actually find anything? */
	if( pair == NULL || pair->key == NULL || strcmp( key, pair->key ) != 0 ) {
		return NULL;

	} else {
		return pair->value;
	}
	
}
void graph_init(graph_t* graph, char* name){
    graph = (graph_t*) malloc(sizeof(graph_t));
    graph->nodes = node_vector_init(0);
    graph->edges = edge_vector_init(0);
    graph->adjacency_map = (adjacency_map*) malloc(sizeof(adjacency_map));

    graph->name = (char*) malloc(strlen(name) * sizeof(char));

    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }
    
    redis_client_set_node_size(graph->redis_context, graph->name, 0);
    redis_client_set_edge_size(graph->redis_context, graph->name, 0);
}

graph_t* graph_load(char* name){
    //First we handle making the actual object
    graph_t* graph;
    hipMallocManaged(&graph, sizeof(graph_t));
    //Then we deal with the interior nodes
    graph->nodes = node_vector_init(0);
    graph->edges = edge_vector_init(0);

    graph->name = (char*) malloc(strlen(name) * sizeof(char));
    graph->name = name;

    //Initialize context to server
    graph->redis_context = redisConnect("127.0.0.1", 6379);

    if (graph->redis_context != NULL && graph->redis_context->err){
        printf("Error initializing redis: %s\n", graph->redis_context->err);
    }

    int node_size =  redis_client_get_node_size(graph->redis_context, graph->name);
    int edge_size = redis_client_get_edge_size(graph->redis_context, graph->name);

    //Get the nodes from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < node_size; i++){
        //node_t* node = node_init_gpu(i);
        //node_load(node, graph->redis_context, graph->name);
        //node_vector_add(graph->nodes, node);
    }
    
    //Get the edges from the database
    #pragma omp parallel for ordered schedule(dynamic)
    for (int i = 0; i < edge_size; i++){
        //edge_t* edge = edge_init_gpu(i, -1, -1);
        //edge_load(edge, graph->redis_context, graph->name);
        //edge_vector_add(graph->edges, edge);
    }

    //graph->adjacency_map = adjacency_map_init(node_size, graph->edges->stuff, edge_size);
    return graph;
}

void graph_add_node(graph_t* graph, node_t* node){
    //node_save(node, graph->redis_context, graph->name);
    //Take care of database stuff. Then add to graph
    redisReply* reply;
    reply = (redisReply*)redisCommand(graph->redis_context, "INCR %s:nodes:size", graph->name);
    freeReplyObject(reply);

    //Now add to graph
    node_vector_add(graph->nodes, node);
}

void graph_add_edge(graph_t* graph, edge_t* edge){
    edge_save(edge, graph->redis_context, graph->name);
    redisReply* reply;
    reply = (redisReply*)redisCommand(graph->redis_context, "INCR %s:edges:size", graph->name);
    freeReplyObject(reply);

    edge_vector_add(graph->edges, edge);
}

void graph_add_edge_between_nodes(graph_t* graph, edge_t* edge, node_t* n_start, node_t* n_end){
    edge->n_start = n_start->id;
    edge->n_end = n_end->id;
    graph_add_edge(graph, edge);
}

__global__ void graph_find(node_vector_t* nodes, char* key, char* value, node_vector_t* d_node_out){
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    max(i, 0);
    min(i, nodes->size);

    node_t* local_node = nodes->stuff[i];
    char* holder = ht_get(local_node->attr, key);

    if ( holder != NULL && holder == value ){
	node_t** vec2 = (node_t**)malloc(sizeof(node_t*) * (nodes->size + 1));
	memcpy(vec2, nodes->stuff, sizeof(node_t*) * (nodes->size));
	free(nodes->stuff);
	nodes->stuff = vec2;
	nodes->size++;
	nodes->stuff[nodes->size - 1] = local_node;
    }
}

node_vector_t* graph_find_V(graph_t* graph, char* key, char* value){

    char* d_key;
    
    hipMalloc((void**)&d_key, strlen(key) * sizeof(char));
    
    hipMemcpy(d_key, key, strlen(key) * sizeof(char), hipMemcpyHostToDevice);
    node_vector_t* d_node_out = node_vector_init(0);
    graph_find<<<1, 1024>>>(graph->nodes, key, value, d_node_out);

    hipFree(d_key);
    return d_node_out;
}
