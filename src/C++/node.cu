#include "edge.cuh"
#include <hip/hip_runtime.h>

node_t* node_init_gpu(int _id){
	node_t* e;
	hipMallocManaged(&e, sizeof(node_t));
	
	e->id = _id;
	
	return e;
}


