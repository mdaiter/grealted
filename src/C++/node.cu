#include "node.cuh"
#include<hip/hip_runtime.h>
Node::Node(){
    id = -1;
    edges = new std::vector<int>();
    attributes = new std::map<std::string, std::string>();
}

Node::Node(int _id){
    id = _id;
    edges = new std::vector<int>();
    attributes = new std::map<std::string, std::string>();
}

Node::~Node(){
    delete edges;
    delete attributes;
}

void Node::setAttribute(std::string key, std::string value){
    attributes->operator[](key) = value;
}

void Node::setID(int _id){
    id = _id;
}

void Node::addEdge(int _edge){
    edges->push_back(_edge);
}

void Node::removeEdge(int _edge){

}

void Node::removeAttribute(std::string attr){
    attributes->erase(attr);
}

std::vector<int> Node::getEdges(){
    return *edges;
}

std::string Node::getAttribute(std::string key){
    return attributes->operator[](key);
}

int Node::getID(){
    return id;
}


