#include "edge.cuh"
#include <hip/hip_runtime.h>
#include "../C/node.h"
#include "../C/kvec.h"
node_t* node_init_gpu(int _id){
	node_t* e;
	hipMallocManaged(&e, sizeof(node_t));
	e->attr = ht_create(16);
	//Add sample to hashtable
	ht_set(e->attr, "", "");
	kv_init(e->edges);
	e->id = _id;
	return e;
}


