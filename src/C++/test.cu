#include <hip/hip_runtime.h>
#include "edge_vector.cuh"
#include "edge.cuh"
#include <stdio.h>

int main(){
	edge_vector_t* vec = edge_vector_init(0);
	
	edge_t* n1 = edge_init_gpu(1, 2, 2);
	edge_t* n2 = edge_init_gpu(2, 1, 2);
	
	edge_vector_add(vec, n1);
	edge_vector_add(vec, n2);
	printf("%d %d \n", vec->stuff[0]->id, vec->stuff[1]->id );
	return 0;
}
